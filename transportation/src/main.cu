#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <filesystem>
#include <math.h>
#include <mutex>
#include <vector>   

#include "flamegpu/flamegpu.h"

#include <GL/glew.h>
#include <GL/GL.h>
#include <GL/GLU.h>
#include <GL/freeglut.h>
#include <GL/glut.h>

#include "clustering\fastcluster.h"

// Grid Size ����������� ����������� ������������ � ��������
#define GRID_WIDTH 800
#define GRID_HEIGHT 800

#define TIME_STOP 3600 // ���������� �������� ������� (� ��������, ����� T=3600 = 1 ���)
#define RUN_COUNT 100 // ���������� �������� ��� ��������
#define DEVISOR 1000000. // ���������, �������� �������� ���������� ��� ������������ ����������
# define M_PI           3.14159265358979323846

// Visualisation mode (1=standalone run, 0 = essemble run, 3 - variation experiments)
#define VIS_MODE 1


#define INTERACTION_RADIUS 150 // ������ ��� ������ ���������
#define PERSONAL_RADIUS 30 // ������ ������� ������������ ������, �������������� �������������� ���������� � ���������� ��������


std::atomic<unsigned int> DRN = { 0 }; // ��� ���
std::atomic <unsigned int> L = { 0 }; // ����� �����
std::atomic <unsigned int> w = { 0 }; // ������ �����
std::atomic <unsigned int> N_nodes = { 0 }; // ���������� ������� ���
std::atomic <unsigned int> R1 = { 0 }; // ���������� ������ ���� ��������� ��������
std::atomic <unsigned int> R2 = { 0 }; 
std::atomic <unsigned int> R3 = { 0 };
std::atomic <unsigned int> R4 = { 0 };
std::atomic <unsigned int> R5 = { 0 };

unsigned int indent_x; // ������ �� x
unsigned int indent_y; // ������ �� y 


std::mutex m;

int window_width = 1050;
int window_height = 1050;
int window_id = 0;

const long MAX_AGENTS = 100000; //������������ ���������� �������

float x_a[MAX_AGENTS], y_a[MAX_AGENTS]; //���������� �������
float rot_a[MAX_AGENTS]; // ���� �������� ��
int agent_class[MAX_AGENTS]; // ����� ������ (��� � ���)
float r_a[MAX_AGENTS]; // ������ ������� ������������ �������
unsigned int agent_state[MAX_AGENTS]; // 

int a_size = 0; // ���������� �������

int jam_count = 0; // ���������� �������� �������
int jam_size = 0; // ��������� �������� �������
int density_jam[1000]; // ��������� �������� �������
float x_jam[1000], y_jam[1000]; //���������� �������� �������


std::ofstream out("results.txt", std::ios::app);
std::ofstream out2("log.txt", std::ios::app); // ��� ��� �������� �������

void display(void);

extern void initVisualisation();
extern void runVisualisation();

__shared__ unsigned int agent_nextID;

__host__ __device__  unsigned int getNextID() {
    agent_nextID++;
    return agent_nextID;
}

//������ ������� ������������ ������-��
inline __device__ double new_radius(double density, double gamma)
{
    return  gamma* (PERSONAL_RADIUS / pow(density, 0.2));
}
//��������� ����� ��������
inline __device__ double distance_agents(double x1, double x2, double y1, double y2)
{
	return  pow(pow(x1 - x2, 2) + pow(y1 - y2, 2), 0.5);
}

class Points_ugvs {
public:
    double x;
    double y;
    Points_ugvs(double xx = 0.0, double yy = 0.0) { x = xx; y = yy; }
    Points_ugvs(const Points_ugvs& p) { x = p.x; y = p.y; }
    double norm() { return(sqrt(x * x + y * y)); }
};

// ���������� ��������� ����� ���
__host__ double distance(const  Points_ugvs& s1, const  Points_ugvs& s2) {
    return sqrt(pow(s2.x - s1.x, 2) + pow(s2.y - s1.y, 2));
}

//��������� ��� �������� ������ � ��������� ���
struct CLUSTERS_UGVs {
public:
    double  x_c;
    double  y_c;
    int  d_c;
    CLUSTERS_UGVs(double xx = 0.0, double yy = 0.0, int dc = 0) { x_c = xx; y_c = yy; d_c = dc;  }
};

bool my_clusters_comparison(const CLUSTERS_UGVs& a, const CLUSTERS_UGVs& b)
{
    return a.d_c > b.d_c;
}



GLuint  textura_id1, textura_id2;

struct textura_struct
{
    int W;
    int H;
    unsigned char* Image;
}get_textura;

int LoadTexture(char* FileName, GLuint &textura_id)
{
    FILE* F;
    /* ��������� ���� */
    if ((F = fopen(FileName, "rb")) == NULL)
        return 0;
    /*������������ � bmp-����� �� ������ �������, � ��������� ������ � ������ */
    fseek(F, 18, SEEK_SET);
    fread(&(get_textura.W), 2, 1, F);
    fseek(F, 2, SEEK_CUR);
    fread(&(get_textura.H), 2, 1, F);

    //printf("%d x %d\n", get_textura.W, get_textura.H);

    /* �������� ������ ��� �����������. ���� ������ �� ����������, ��������� ���� � ������� � ������� */
    if ((get_textura.Image = (unsigned char*)malloc(sizeof(unsigned char) * 3 * get_textura.W * get_textura.H)) == NULL)
    {
        fclose(F);
        return 0;
    }
    /* ��������� ����������� � ������ �� 3 ����, �� ���� RGB ��� ������� ������� */
    fseek(F, 30, SEEK_CUR);
    fread(get_textura.Image, 3, get_textura.W * get_textura.H, F);

    glGenTextures(1, &textura_id);
    glBindTexture(GL_TEXTURE_2D, textura_id);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    gluBuild2DMipmaps(GL_TEXTURE_2D, 3, get_textura.W, get_textura.H, GL_RGB, GL_UNSIGNED_BYTE, get_textura.Image);
    free(get_textura.Image);
    fclose(F);

    return 1;
}

int initGL()
{
    // initialize necessary OpenGL extensions
    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 "
        "GL_ARB_pixel_buffer_object"
    )) {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return FALSE;
    }
    // default initialization
    glClearColor(1.0, 1.0, 1.0, 1.0);
  
}

void reshape(int w, int h)
{
    glViewport(0, 0, w, h);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, w, 0, h);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

void windowResize(int width, int height) {
    window_width = width;
    window_height = height;
}

extern void initVisualisation()
{
    // Create GL context
    int   argc = 1;
    char glutString[] = "GLUT application";
    char* argv[] = { glutString, NULL };
    //char *argv[] = {"GLUT application", NULL};	

    glutInit(&argc, argv);


    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
    glutInitWindowSize(window_width, window_height);
    window_id = glutCreateWindow("FLAME GPU Visualiser");
    glutReshapeFunc(windowResize);

    // initialize GL
    if (FALSE == initGL()) {
        return;
    }

    if (LoadTexture((char*)"mgv.bmp", textura_id1) != 1) { printf("�� ������� ��������� �����������\n"); }
    if (LoadTexture((char*)"ugv.bmp", textura_id2) != 1) { printf("�� ������� ��������� �����������\n"); }

    glutDisplayFunc(display);
    glutReshapeFunc(reshape);

    //����� ����� ��������������������
}

extern void runVisualisation()
{
    // start rendering mainloop
    glutMainLoop();


}

//������� ��������� ����������
void drawCircle(float x, float y, float r, int amountSegments)
{
    glColor3d(0, 0, 0);
    glLineWidth(2.0f);

    glEnable(GL_LINE_SMOOTH);
    glHint(GL_LINE_SMOOTH_HINT, GL_NICEST);
    glEnable(GL_POINT_SMOOTH);
    glHint(GL_POINT_SMOOTH_HINT, GL_NICEST);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    glBegin(GL_LINE_LOOP);
    
    for (int i = 0; i < amountSegments; i++)
    {
        float angle = 2.0 * 3.1415926 * float(i) / float(amountSegments);
        float dx = r * cosf(angle);
        float dy = r * sinf(angle);
        glVertex2f(x + dx, y + dy);
    }
    glEnd();
}

void drawCircle2(float x, float y, float r, int amountSegments)
{
    glColor3d(0, 0, 0);

    glEnable(GL_LINE_SMOOTH);
    glHint(GL_LINE_SMOOTH_HINT, GL_NICEST);
    glEnable(GL_POINT_SMOOTH);
    glHint(GL_POINT_SMOOTH_HINT, GL_NICEST);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);


    glPushAttrib(GL_ENABLE_BIT);
    glLineWidth(1);
    glLineStipple(1, 0xAAAA);
    glEnable(GL_LINE_STIPPLE);
    glBegin(GL_LINE_LOOP);

    for (int i = 0; i < amountSegments; i++)
    {
        float angle = 2.0 * 3.1415926 * float(i) / float(amountSegments);
        float dx = r * cosf(angle);
        float dy = r * sinf(angle);
        glVertex2f(x + dx, y + dy);
    }
    glEnd();
    glDisable(GL_LINE_STIPPLE);
}

void drawCircle3(float x, float y, float r, int pointCount, int level)
{

    /*
    if (level == 1)
    glColor4f(0.0, 1.0, 0.0, 0.5); //�������
    if (level == 2)
    glColor4f(1.0, 1.0, 0.0, 0.5); //������
    if (level == 3)
    glColor4f(1.0, 0.0, 0.0, 0.5); //�������
    */

    if (level == 1)
        glColor4f(0.2, 0.2, 0.2, 0.5); //�����
    if (level == 2)
        glColor4f(0.2, 0.2, 0.2, 0.75); //�����
    if (level == 3)
        glColor4f(0.2, 0.2, 0.2, 0.9); //�����

    glEnable(GL_LINE_SMOOTH);
    glHint(GL_LINE_SMOOTH_HINT, GL_NICEST);
    glEnable(GL_POINT_SMOOTH);
    glHint(GL_POINT_SMOOTH_HINT, GL_NICEST);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    glPushAttrib(GL_ENABLE_BIT);
    glLineWidth(4);
    glLineStipple(1, 0xAAAA);
    glEnable(GL_LINE_STIPPLE);

    glBegin(GL_TRIANGLE_FAN);

    const float step = float(2 * M_PI) / pointCount;
    for (float angle = 0; angle <= float(2 * M_PI); angle += step)
    {
        float a = (fabsf(angle - float(2 * M_PI)) < 0.00001f) ? 0.f : angle;
        const float dx = r * cosf(a);
        const float dy = r * sinf(a);
        glVertex2f(dx + x, dy + y);
    }
    glEnd();
    glDisable(GL_LINE_STIPPLE);
}

void display()
{
    glutSetWindow(window_id);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    //������������ �������� �������� ����

    

    if (DRN == 1) // ���������-��������� � ����� ����� ��������� ��������
    {
        /*
        double L = 1000; // ����� �����
        double w = 100; // ������ �����
        double R1 = 300;
        double R2 = R1 + w / 2;
        double R3 = R2 + w / 2;
        */

        // ������ ������ (��������������)************************************************************
        glLineWidth(2);       // ������ �����
        glBegin(GL_LINES);
        glColor3d(0.0, 0.0, 0.0);     // ������ ����

        //������� ������ ������
        glVertex2f(indent_x, indent_y + L/2 + w);
        glVertex2f(indent_x + L, indent_y + L/2 + w);

        //�������� ��������������
        glVertex2f(indent_x, indent_y + L/2 );
        glVertex2f(indent_x + L, indent_y + L/2);

        //������� ������ ������
        glVertex2f(indent_x, indent_y + L/2 - w);
        glVertex2f(indent_x + L, indent_y + L/2 - w);

        glEnd();

        //���������� ������ ������
        glPushAttrib(GL_ENABLE_BIT);
        glLineWidth(1);
        glLineStipple(1, 0xAAAA);
        glEnable(GL_LINE_STIPPLE);
        glBegin(GL_LINES);
        glVertex2f(indent_x, indent_y + L / 2 + w / 2);
        glVertex2f(indent_x + L, indent_y + L / 2 + w / 2);

        glVertex2f(indent_x, indent_y + L / 2 - w / 2);
        glVertex2f(indent_x + L, indent_y + L / 2 - w / 2);

        glEnd();
        glDisable(GL_LINE_STIPPLE);

        // ������ ������ (������������)**********************************************************************************

        glLineWidth(2);       // ������ �����
        glBegin(GL_LINES);
        glColor3d(0.0, 0.0, 0.0);     // ������ ����

        //������� ������ ������
        glVertex2f(indent_x + L / 2 - w, indent_y);
        glVertex2f(indent_x + L / 2 - w, indent_y + L);

        //�������� ��������������
        glVertex2f(indent_x + L / 2, indent_y);
        glVertex2f(indent_x + L / 2, indent_y + L);

        //������� ������ ������
        glVertex2f(indent_x + L / 2 + w, indent_y);
        glVertex2f(indent_x + L / 2 + w, indent_y + L);

        glEnd();

        //���������� ������ ������
        glPushAttrib(GL_ENABLE_BIT);
        glLineWidth(1);
        glLineStipple(1, 0xAAAA);
        glEnable(GL_LINE_STIPPLE);
        glBegin(GL_LINES);
        glVertex2f(indent_x + L/2 - w / 2, indent_y);
        glVertex2f(indent_x + L / 2 - w / 2, indent_y + L);

        glVertex2f(indent_x + L / 2 + w / 2, indent_y);
        glVertex2f(indent_x + L / 2 + w / 2, indent_y + L);

        glEnd();
        glDisable(GL_LINE_STIPPLE);


        //���� ��������� ��������*********************************************
        //���������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R1, 5000);
        //���������� �� ������
        drawCircle2(indent_x + L / 2, indent_y + L / 2, R2, 5000);
        //������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R3, 5000);

    }

    if (DRN == 2) // ���������-��������� � ����� ������ ��������� ��������
    {
        /*
        double L = 1000; // ����� �����
        double w = 100; // ������ �����
        double R1 = 250;
        double R2 = R1 + w / 2;
        double R3 = R2 + w / 2;
        double R4 = R3 + w / 2;
        double R5 = R4 + w / 2;
        */
        // ������ ������ (��������������)************************************************************
        glLineWidth(2);       // ������ �����
        glBegin(GL_LINES);
        glColor3d(0.0, 0.0, 0.0);     // ������ ����

        //������� ������ ������
        glVertex2f(indent_x, indent_y + L / 2 + w);
        glVertex2f(indent_x + L, indent_y + L / 2 + w);

        //�������� ��������������
        glVertex2f(indent_x, indent_y + L / 2);
        glVertex2f(indent_x + L, indent_y + L / 2);

        //������� ������ ������
        glVertex2f(indent_x, indent_y + L / 2 - w);
        glVertex2f(indent_x + L, indent_y + L / 2 - w);

        glEnd();

        //���������� ������ ������
        glPushAttrib(GL_ENABLE_BIT);
        glLineWidth(1);
        glLineStipple(1, 0xAAAA);
        glEnable(GL_LINE_STIPPLE);
        glBegin(GL_LINES);
        glVertex2f(indent_x, indent_y + L / 2 + w / 2);
        glVertex2f(indent_x + L, indent_y + L / 2 + w / 2);

        glVertex2f(indent_x, indent_y + L / 2 - w / 2);
        glVertex2f(indent_x + L, indent_y + L / 2 - w / 2);

        glEnd();
        glDisable(GL_LINE_STIPPLE);

        // ������ ������ (������������)**********************************************************************************

        glLineWidth(2);       // ������ �����
        glBegin(GL_LINES);
        glColor3d(0.0, 0.0, 0.0);     // ������ ����

        //������� ������ ������
        glVertex2f(indent_x + L / 2 - w, indent_y);
        glVertex2f(indent_x + L / 2 - w, indent_y + L);

        //�������� ��������������
        glVertex2f(indent_x + L / 2, indent_y);
        glVertex2f(indent_x + L / 2, indent_y + L);

        //������� ������ ������
        glVertex2f(indent_x + L / 2 + w, indent_y);
        glVertex2f(indent_x + L / 2 + w, indent_y + L);

        glEnd();

        //���������� ������ ������
        glPushAttrib(GL_ENABLE_BIT);
        glLineWidth(1);
        glLineStipple(1, 0xAAAA);
        glEnable(GL_LINE_STIPPLE);
        glBegin(GL_LINES);
        glVertex2f(indent_x + L / 2 - w / 2, indent_y);
        glVertex2f(indent_x + L / 2 - w / 2, indent_y + L);

        glVertex2f(indent_x + L / 2 + w / 2, indent_y);
        glVertex2f(indent_x + L / 2 + w / 2, indent_y + L);

        glEnd();
        glDisable(GL_LINE_STIPPLE);


        //���� ��������� ��������*********************************************
        //���������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R1, 5000);
        //���������� �� ������
        drawCircle2(indent_x + L / 2, indent_y + L / 2, R2, 5000);
        //������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R3, 5000);
        //���������� �� ������
        drawCircle2(indent_x + L / 2, indent_y + L / 2, R4, 5000);
        //������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R5, 5000);

    }
    
    if (DRN == 3) // �������������
    {
        /*
        double L = 1000; // ����� �����
        double w = 100; // ������ �����
        int N_nodes = 2; // ���������� ������� ���
        */

        // �������������� ������ ************************************************************
        double y_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, indent_y + y_initial + L / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial + 2 * w;

        }

        y_initial = 0;
        for (int i = 0; i < N_nodes; i++)

        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, indent_y + y_initial + L / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial - 2 * w;

        }

        // ������������ ������ **********************************************************************************
        double x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial + 2 * w;
        }
        
        x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial - 2 * w;
        }


    }

    if (DRN == 4) // ������������-������������
    {
        /*
        double L = 1000; // ����� �����
        double w = 100; // ������ �����
        int N_nodes = 2; // ���������� ������� ���
        */


        // �������������� ������ ************************************************************
        double y_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial +  L / 2 + w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, indent_y + y_initial + L / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial + 2 * w;

        }


        y_initial = 0;
        for (int i = 0; i < N_nodes; i++)

        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, indent_y + y_initial + L / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial - 2 * w;

        }
       
        // ������������ ������ **********************************************************************************
       
        double x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial + 2 * w;
        }

        x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial - 2 * w;
        }
        

        // ���������******************************************************************

        glLineWidth(2);       // ������ �����
        glBegin(GL_LINES);
        glColor3d(0.0, 0.0, 0.0);     // ������ ����

        // �����-�������
        //������� ������ ������
        glVertex2f(indent_x + (double)w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L + (double)w / sin(45 * M_PI / 180), indent_y);

        //�������� �������������� �����
        glVertex2f(indent_x, indent_y + L);
        glVertex2f(indent_x + L, indent_y);

        //������� ������ ������
        glVertex2f(indent_x - (double) w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L - (double) w / sin(45 * M_PI / 180), indent_y);

        // ������-������
        
        //������� ������ ������
        glVertex2f(indent_x + L - (double)w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x - (double)w / sin(45 * M_PI / 180), indent_y);

        //�������� �������������� �����
        glVertex2f(indent_x + L, indent_y + L);
        glVertex2f(indent_x, indent_y);

        //������� ������ ������
        glVertex2f(indent_x + L + (double)w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + (double)w / sin(45 * M_PI / 180), indent_y);

        glEnd();


        //���������� ������ ������
        glPushAttrib(GL_ENABLE_BIT);
        glLineWidth(1);
        glLineStipple(1, 0xAAAA);
        glEnable(GL_LINE_STIPPLE);
        glBegin(GL_LINES);

        // �����-�������
        glVertex2f(indent_x + (double) (w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L + (double)(w / 2) / sin(45 * M_PI / 180), indent_y);

        glVertex2f(indent_x - (double) (w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L - (double) (w / 2) / sin(45 * M_PI / 180), indent_y);

        // ������-������
        glVertex2f(indent_x + L - (double) (w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x - (double) (w / 2) / sin(45 * M_PI / 180), indent_y);

        glVertex2f(indent_x + L + (double) (w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + (double) (w / 2) / sin(45 * M_PI / 180), indent_y);

        glEnd();
    }

    if (DRN == 5) // �����������-���������
    {
        /*
        double L = 1000; // ����� �����
        double w = 100; // ������ �����
        double R1 = 200;
        double R2 = R1 + w / 2;
        double R3 = R2 + w / 2;
        double R4 = R3 + w / 2;
        double R5 = R4 + w / 2;
        int N_nodes = 2; // ���������� ������� ���
        */

        // �������������� ������ ************************************************************
        double y_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, y_initial + L / 2 + w);
            glVertex2f(indent_x + L, y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, y_initial + L / 2);
            glVertex2f(indent_x + L, y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, y_initial + L / 2 - w);
            glVertex2f(indent_x + L, y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial + 2 * w;

        }

        y_initial = 0;
        for (int i = 0; i < N_nodes; i++)

        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, y_initial + L / 2 + w);
            glVertex2f(indent_x + L, y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, y_initial + L / 2);
            glVertex2f(indent_x + L, y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, y_initial + L / 2 - w);
            glVertex2f(indent_x + L, y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial - 2 * w;

        }

        // ������������ ������ **********************************************************************************
        double x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial + 2 * w;
        }

        x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial - 2 * w;
        }

        //���� ��������� ��������*********************************************
        //���������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R1, 5000);
        //���������� �� ������
        drawCircle2(indent_x + L / 2, indent_y + L / 2, R2, 5000);
        //������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R3, 5000);
        //���������� �� ������
        drawCircle2(indent_x + L / 2, indent_y + L / 2, R4, 5000);
        //������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R5, 5000);


    }
     
    if (DRN == 6) // ���������������: ������������-�����������-���������
    {
        /*
        double L = 1000; // ����� �����
        double w = 100; // ������ �����
        int N_nodes = 2; // ���������� ������� ���
        double R1 = 200;
        double R2 = R1 + w / 2;
        double R3 = R2 + w / 2;
        double R4 = R3 + w / 2;
        double R5 = R4 + w / 2;
        */

        // �������������� ������ ************************************************************
        double y_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, indent_y + y_initial + L / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial + 2 * w;

        }


        y_initial = 0;
        for (int i = 0; i < N_nodes; i++)

        {

            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w);

            //�������� ��������������
            glVertex2f(indent_x, indent_y + y_initial + L / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2);

            //������� ������ ������
            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x, indent_y + y_initial + L / 2 + w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 + w / 2);

            glVertex2f(indent_x, indent_y + y_initial + L / 2 - w / 2);
            glVertex2f(indent_x + L, indent_y + y_initial + L / 2 - w / 2);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            y_initial = y_initial - 2 * w;

        }

        // ������������ ������ **********************************************************************************

        double x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial + 2 * w;
        }

        x_initial = 0;
        for (int i = 0; i < N_nodes; i++)
        {
            glLineWidth(2);       // ������ �����
            glBegin(GL_LINES);
            glColor3d(0.0, 0.0, 0.0);     // ������ ����

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w, indent_y + L);

            //�������� ��������������
            glVertex2f(indent_x + x_initial + L / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2, indent_y + L);

            //������� ������ ������
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w, indent_y + L);

            glEnd();

            //���������� ������ ������
            glPushAttrib(GL_ENABLE_BIT);
            glLineWidth(1);
            glLineStipple(1, 0xAAAA);
            glEnable(GL_LINE_STIPPLE);
            glBegin(GL_LINES);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 - w / 2, indent_y + L);

            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y);
            glVertex2f(indent_x + x_initial + L / 2 + w / 2, indent_y + L);

            glEnd();
            glDisable(GL_LINE_STIPPLE);

            x_initial = x_initial - 2 * w;
        }


        // ���������******************************************************************

        glLineWidth(2);       // ������ �����
        glBegin(GL_LINES);
        glColor3d(0.0, 0.0, 0.0);     // ������ ����

        // �����-�������
        //������� ������ ������
        glVertex2f(indent_x + (double)w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L + (double)w / sin(45 * M_PI / 180), indent_y);

        //�������� �������������� �����
        glVertex2f(indent_x, indent_y + L);
        glVertex2f(indent_x + L, indent_y);

        //������� ������ ������
        glVertex2f(indent_x - (double)w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L - (double)w / sin(45 * M_PI / 180), indent_y);

        // ������-������

        //������� ������ ������
        glVertex2f(indent_x + L - (double)w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x - (double)w / sin(45 * M_PI / 180), indent_y);

        //�������� �������������� �����
        glVertex2f(indent_x + L, indent_y + L);
        glVertex2f(indent_x, indent_y);

        //������� ������ ������
        glVertex2f(indent_x + L + (double)w / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + (double)w / sin(45 * M_PI / 180), indent_y);

        glEnd();


        //���������� ������ ������
        glPushAttrib(GL_ENABLE_BIT);
        glLineWidth(1);
        glLineStipple(1, 0xAAAA);
        glEnable(GL_LINE_STIPPLE);
        glBegin(GL_LINES);

        // �����-�������
        glVertex2f(indent_x + (double)(w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L + (double)(w / 2) / sin(45 * M_PI / 180), indent_y);

        glVertex2f(indent_x - (double)(w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + L - (double)(w / 2) / sin(45 * M_PI / 180), indent_y);

        // ������-������
        glVertex2f(indent_x + L - (double)(w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x - (double)(w / 2) / sin(45 * M_PI / 180), indent_y);

        glVertex2f(indent_x + L + (double)(w / 2) / sin(45 * M_PI / 180), indent_y + L);
        glVertex2f(indent_x + (double)(w / 2) / sin(45 * M_PI / 180), indent_y);

        glEnd();


        //���� ��������� ��������*********************************************
        //���������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R1, 5000);
        //���������� �� ������
        drawCircle2(indent_x + L / 2, indent_y + L / 2, R2, 5000);
        //������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R3, 5000);
        //���������� �� ������
        drawCircle2(indent_x + L / 2, indent_y + L / 2, R4, 5000);
        //������� ������
        drawCircle(indent_x + L / 2, indent_y + L / 2, R5, 5000);


    }
    
    for (int i = 0; i < a_size; i++)
    {
        glPointSize(4);

       // glBegin(GL_POINTS);

               
        if (agent_class[i] == 1 && agent_state[i] != 1) // ���
        {
            /* ����� ����������� � ���� */
            glPushMatrix();
          
            glEnable(GL_TEXTURE_2D);
            glColor3f(1, 1, 1);

            glBindTexture(GL_TEXTURE_2D, textura_id1); // ���

            glTranslatef(x_a[i], y_a[i], 0);
            glRotatef(rot_a[i] * 180 / M_PI, 0.0f, 0.0f, 1.0f); //������� ��
            glTranslatef(-x_a[i], -y_a[i], 0);

            glBegin(GL_QUADS);
            glTexCoord2d(0, 0); glVertex2d( (x_a[i] - 15), (y_a[i] - 5) );
            glTexCoord2d(0, 1); glVertex2d( (x_a[i] - 15), (y_a[i] + 5) );
            glTexCoord2d(1, 1); glVertex2d( (x_a[i] + 15), (y_a[i] + 5) );
            glTexCoord2d(1, 0); glVertex2d( (x_a[i] + 15), (y_a[i] - 5) );
            glEnd();
            
            glDisable(GL_TEXTURE_2D);
            glPopMatrix();
        }

        if (agent_class[i] == 2 && agent_state[i] != 1) // ���
        {
            /* ����� ����������� � ���� */
            glPushMatrix();
            
            glEnable(GL_TEXTURE_2D);
            glColor3f(1, 1, 1);

            glBindTexture(GL_TEXTURE_2D, textura_id2); // ���
            
            glTranslatef(x_a[i], y_a[i], 0);
            glRotatef(rot_a[i] * 180/ M_PI, 0.0f, 0.0f, 1.0f); //������� ��
            glTranslatef(-x_a[i], -y_a[i], 0);

            glBegin(GL_QUADS);
            glTexCoord2d(0, 0); glVertex2d(x_a[i] - 15, y_a[i] - 5);
            glTexCoord2d(0, 1); glVertex2d(x_a[i] - 15, y_a[i] + 5);
            glTexCoord2d(1, 1); glVertex2d(x_a[i] + 15, y_a[i] + 5);
            glTexCoord2d(1, 0); glVertex2d(x_a[i] + 15, y_a[i] - 5);
            glEnd();

            glDisable(GL_TEXTURE_2D);
            glPopMatrix();
           
        }

        if (agent_state[i] == 1) // �� � ��������� ���������
        {
            /* ����� ����������� � ���� */
            glPushMatrix();

            glEnable(GL_TEXTURE_2D);
            glColor3f(0, 0, 0);

            glBindTexture(GL_TEXTURE_2D, textura_id2); // ���

            glTranslatef(x_a[i], y_a[i], 0);
            glRotatef(rot_a[i] * 180 / M_PI, 0.0f, 0.0f, 1.0f); //������� ��
            glTranslatef(-x_a[i], -y_a[i], 0);

            glBegin(GL_QUADS);
            glTexCoord2d(0, 0); glVertex2d(x_a[i] - 15, y_a[i] - 5);
            glTexCoord2d(0, 1); glVertex2d(x_a[i] - 15, y_a[i] + 5);
            glTexCoord2d(1, 1); glVertex2d(x_a[i] + 15, y_a[i] + 5);
            glTexCoord2d(1, 0); glVertex2d(x_a[i] + 15, y_a[i] - 5);
            glEnd();

            glDisable(GL_TEXTURE_2D);
            glPopMatrix();
        }

        //glVertex3d(x_a[i], y_a[i], 0); // ������-�����
        //glEnd();

        drawCircle2(x_a[i], y_a[i], r_a[i], 100); // ������ ������� ������������ ��

    }

       
    //������������ "������"
    for (int i = 0; i < jam_count; i++)
    {
        //��������� ���������� ���������
        glPointSize(10);
        glBegin(GL_POINTS);
        glColor4f(1.0f, 0.5f, 0.0f, 0.0f);//orange/brown
        glVertex3d(x_jam[i], y_jam[i], 0); // ������ ���������
        glEnd();

        int lvl = 1;
        if (density_jam[i] < 6)
            lvl = 1;
        if (density_jam[i] >= 6 && density_jam[i] < 10)
            lvl = 2;
        if (density_jam[i] >= 10)
            lvl = 3;

        drawCircle3(x_jam[i], y_jam[i], PERSONAL_RADIUS *3, 100, lvl); 

    }

  
    
    
    //redraw

glutSwapBuffers();
glutPostRedisplay();

}


void timer(int = 0)
{
    display();
    glutTimerFunc(1, timer, 0);
}

std::atomic_int par1; //��������� ������������ ������ ���
std::atomic_int par2;
std::atomic_int objective; //������� ������� ���


FLAMEGPU_INIT_FUNCTION(init_function) {
    std::lock_guard<std::mutex> lock(m);

    //�������� ������������ ���*************************************************************************
    DRN = FLAMEGPU->environment.getProperty<unsigned int>("DRN");
    L = FLAMEGPU->environment.getProperty<unsigned int>("L");
    w = FLAMEGPU->environment.getProperty<unsigned int>("w");
    N_nodes = FLAMEGPU->environment.getProperty<unsigned int>("N_nodes");
    R1 = FLAMEGPU->environment.getProperty<unsigned int>("R1");
    R2 = R1 + w / 2;
    R3 = R2 + w / 2;
    R4 = R3 + w / 2;
    R5 = R4 + w / 2;


    indent_x= FLAMEGPU->environment.getProperty<unsigned int>("indent_x"); // ������� �� x � y ��� ���
    indent_y = FLAMEGPU->environment.getProperty<unsigned int>("indent_y");
}

FLAMEGPU_STEP_FUNCTION(BasicOutput) {
    std::lock_guard<std::mutex> lock(m);

    flamegpu::HostAgentAPI agent = FLAMEGPU->agent("agent-vehicles");

    DRN = FLAMEGPU->environment.getProperty<unsigned int>("DRN");
    
    //��������� �������-��� � ���������� �� � ���
    int direction = 0;
    double x = 0;
    double y = 0;
    double x2 = 0;
    double y2 = 0;
    double x3 = 0;
    double y3 = 0;
    double v = 0;

    int x_min = indent_x;
    int x_max = x_min;
    int y_min = indent_y + L / 2;
    int y_max = indent_y + L / 2 + w;

    int x_min2 = indent_x;
    int x_max2 = x_min2;
    int y_min2 = indent_y + L / 2 - 2 * w;
    int y_max2 = indent_y + L / 2 - w;

    int x_min3 = indent_x;
    int x_max3 = x_min3;
    int y_min3 = indent_y + L / 2 + 2 * w;
    int y_max3 = indent_y + L / 2 + 3 * w;


    int x_min4 = indent_x;
    int x_max4 = x_min4 + (double) w * cos(45 * M_PI / 180);
    int y_min4 = indent_y;
    int y_max4 = indent_y + (double)w * sin(45 * M_PI / 180);

    int unsigned intensity_of_UGVs = FLAMEGPU->environment.getProperty<unsigned int>("intensity_of_UGVs");

    if (VIS_MODE == 3)
        intensity_of_UGVs = par1;

    if (FLAMEGPU->getStepCounter() % FLAMEGPU->environment.getProperty<unsigned int>("frequency") == 0)
    {
        int n = 0;
        while (n < intensity_of_UGVs)
        {
            
            if (DRN == 1 || DRN == 2 || DRN == 3 ||  DRN == 5)
            direction = FLAMEGPU->random.uniform<int>(1, 4);

            if (DRN == 4 || DRN == 6)
            direction = FLAMEGPU->random.uniform<int>(1, 8);


            switch (direction) {
            case 1: // �����-�������
                x_min = indent_x;
                x_max = x_min;
                y_min = indent_y + L / 2 - w;
                y_max = indent_y + L / 2;

                if(DRN==3 || DRN == 4 || DRN == 5 || DRN == 6)
                    { 
                    x_min2 = indent_x;
                    x_max2 = x_min2;
                    y_min2 = indent_y + L / 2 - 3 * w;
                    y_max2 = indent_y + L / 2 - 2 * w;

                    x_min3 = indent_x;
                    x_max3 = x_min3;
                    y_min3 = indent_y + L / 2 + w;
                    y_max3 = indent_y + L / 2 + 2 * w;
                    }

                break;
            case 2: // ������-������
                x_min = indent_x + L;
                x_max = x_min;

                y_min = indent_y + L / 2;
                y_max = indent_y + L / 2 + w;

                if (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6)
                {
                    x_min2 = indent_x + L;
                    x_max2 = x_min2;
                    y_min2 = indent_y + L / 2 - 2 * w;
                    y_max2 = indent_y + L / 2 - w;

                    x_min3 = indent_x + L;
                    x_max3 = x_min3;
                    y_min3 = indent_y + L / 2 + 2 * w;
                    y_max3 = indent_y + L / 2 + 3 * w;
                }

                break;
            case 3:  // �����-�����
                x_min = indent_x + L / 2;
                x_max = indent_x + L / 2 + w;
                y_min = indent_y;
                y_max = y_min;

                if (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6)
                {
                    x_min2 = indent_x + L / 2 - 2 * w;
                    x_max2 = indent_x + L / 2 - w;
                    y_min2 = indent_y;
                    y_max2 = y_min2;

                    x_min3 = indent_x + L / 2 + 2 * w;
                    x_max3 = indent_x + L / 2 + 3 * w;
                    y_min3 = indent_y;
                    y_max3 = y_min3;
                }

                break;
            case 4: // ������-����
                x_min = indent_x + L / 2 - w;
                x_max = indent_x + L / 2;
                y_min = indent_y + L;
                y_max = y_min;

                if (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6)
                {
                    x_min2 = indent_x + L / 2 - 3 * w;
                    x_max2 = indent_x + L / 2 - 2 * w;
                    y_min2 = indent_y + L;
                    y_max2 = y_min2;

                    x_min3 = indent_x + L / 2 + w;
                    x_max3 = indent_x + L / 2 + 2 * w;
                    y_min3 = indent_y + L;
                    y_max3 = y_min3;
                }

                break;
            case 5: // ���������  �-�
                   x_min = indent_x;
                   x_max = x_min + (double)w * cos (45 * M_PI / 180);
                   y_min = indent_y + w;
                   y_max = y_min + (double)w * sin(45 * M_PI / 180);
               
                   break;
            case 6: // ���������  �-�
                   x_min = indent_x + L / 2 + 4 * w;
                   x_max = x_min + (double)w * cos(45 * M_PI / 180);
                   y_min = indent_y + L / 2 + 4 * w;
                   y_max = y_min + (double)w * sin(45 * M_PI / 180);

                break;

            case 7: // ���������  �-�
                x_min = indent_x;
                x_max = x_min + (double)w * cos(45 * M_PI / 180);
                y_min = indent_y + L / 2 + 4 * w;
                y_max = y_min + (double)w * sin(45 * M_PI / 180);

                break;

            case 8: // ���������  �-�
                x_min = indent_x + L / 2 + 4 * w;
                x_max = x_min + (double)w * cos(45 * M_PI / 180);
                y_min = indent_y + w;
                y_max = y_min + (double)w * sin(45 * M_PI / 180);

                break;
            }
            
            
            x = FLAMEGPU->random.uniform(x_min, x_max);
            y = FLAMEGPU->random.uniform(y_min, y_max);

            
            // �� ��� ������ � ������ ����������� (�.�., �����-�������, ������-������, ������-���� � �.�.)
            if ( (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6) && (direction==1 || direction == 2 || direction == 3 || direction == 4) )
            {
                int share = FLAMEGPU->random.uniform<int>(1, 3);
                x2 = FLAMEGPU->random.uniform(x_min2, x_max2);
                y2 = FLAMEGPU->random.uniform(y_min2, y_max2);
                x3 = FLAMEGPU->random.uniform(x_min3, x_max3);
                y3 = FLAMEGPU->random.uniform(y_min3, y_max3);

                if (share == 1)
                {
                    x = x;
                    y = y;
                }
                if (share == 2)
                {
                    x = x2;
                    y = y2;
                }
                if (share == 3)
                {
                    x = x3;
                    y = y3;
                }
            }

            float vv = FLAMEGPU->environment.getProperty<float>("velocity_of_UGVs");

            if(VIS_MODE==3)
                vv = par2;

            v = FLAMEGPU->random.logNormal<float>(log(vv), 0.1); // ��������� �������� ���
           
            flamegpu::HostNewAgentAPI instance = agent.newAgent();
            instance.setVariable<int>("id", getNextID());
            instance.setVariable<float>("x", x);
            instance.setVariable<float>("y", y);
            instance.setVariable<float>("Ra", PERSONAL_RADIUS);
            instance.setVariable<int>("agent_class", 1);
            instance.setVariable<int>("agent_state", 2);
            instance.setVariable<int>("agent_type", direction);
            instance.setVariable<float>("velocity", v);
            instance.setVariable<float>("rotation", 0);
            instance.setVariable<float>("neighbour_distance", 10000);
            instance.setVariable<float>("other_neighbour_distance", 10000);
            n++;
        }

        n = 0;
        while (n < FLAMEGPU->environment.getProperty<unsigned int>("intensity_of_MGVs"))
        {
            if (DRN == 1 || DRN == 2 || DRN == 3 || DRN == 5)
                direction = FLAMEGPU->random.uniform<int>(1, 4);

            if (DRN == 4 || DRN == 6)
                direction = FLAMEGPU->random.uniform<int>(1, 8);

            switch (direction) {
            case 1: // �����-�������
                x_min = indent_x;
                x_max = x_min;
                y_min = indent_y + L / 2 - w;
                y_max = indent_y + L / 2;

                if (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6)
                {
                    x_min2 = indent_x;
                    x_max2 = x_min2;
                    y_min2 = indent_y + L / 2 - 3 * w;
                    y_max2 = indent_y + L / 2 - 2 * w;

                    x_min3 = indent_x;
                    x_max3 = x_min3;
                    y_min3 = indent_y + L / 2 + w;
                    y_max3 = indent_y + L / 2 + 2 * w;
                }

                break;
            case 2: // ������-������
                x_min = indent_x + L;
                x_max = x_min;

                y_min = indent_y + L / 2;
                y_max = indent_y + L / 2 + w;

                if (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6)
                {
                    x_min2 = indent_x + L;
                    x_max2 = x_min2;
                    y_min2 = indent_y + L / 2 - 2 * w;
                    y_max2 = indent_y + L / 2 - w;

                    x_min3 = indent_x + L;
                    x_max3 = x_min3;
                    y_min3 = indent_y + L / 2 + 2 * w;
                    y_max3 = indent_y + L / 2 + 3 * w;
                }

                break;
            case 3:  // �����-�����
                x_min = indent_x + L / 2;
                x_max = indent_x + L / 2 + w;
                y_min = indent_y;
                y_max = y_min;

                if (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6)
                {
                    x_min2 = indent_x + L / 2 - 2 * w;
                    x_max2 = indent_x + L / 2 - w;
                    y_min2 = indent_y;
                    y_max2 = y_min2;

                    x_min3 = indent_x + L / 2 + 2 * w;
                    x_max3 = indent_x + L / 2 + 3 * w;
                    y_min3 = indent_y;
                    y_max3 = y_min3;
                }

                break;
            case 4: // ������-�����
                x_min = indent_x + L / 2 - w;
                x_max = indent_x + L / 2;
                y_min = indent_y + L;
                y_max = y_min;

                if (DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6)
                {
                    x_min2 = indent_x + L / 2 - 3 * w;
                    x_max2 = indent_x + L / 2 - 2 * w;
                    y_min2 = indent_y + L;
                    y_max2 = y_min2;

                    x_min3 = indent_x + L / 2 + w;
                    x_max3 = indent_x + L / 2 + 2 * w;
                    y_min3 = indent_y + L;
                    y_max3 = y_min3;
                }

                break;

            case 5: // ���������  �-�
                x_min = indent_x;
                x_max = x_min + (double)w * cos(45 * M_PI / 180);
                y_min = indent_y + w;
                y_max = y_min + (double)w * sin(45 * M_PI / 180);

                break;
            case 6: // ���������  �-�
                x_min = indent_x + L / 2 + 5 * w;
                x_max = x_min + (double)w * cos(45 * M_PI / 180);
                y_min = indent_y + L / 2 + 4 * w;
                y_max = y_min + (double)w * sin(45 * M_PI / 180);

                break;

            case 7: // ���������  �-�
                x_min = indent_x;
                x_max = x_min + (double)w * cos(45 * M_PI / 180);
                y_min = indent_y + L / 2 + 4 * w;
                y_max = y_min + (double)w * sin(45 * M_PI / 180);

                break;

            case 8: // ���������  �-�
                x_min = indent_x + L / 2 + 4 * w;
                x_max = x_min + (double)w * cos(45 * M_PI / 180);
                y_min = indent_y + w;
                y_max = y_min + (double)w * sin(45 * M_PI / 180);

                break;

            }

                      
            x = FLAMEGPU->random.uniform(x_min, x_max);
            y = FLAMEGPU->random.uniform(y_min, y_max);

            // �� ��� ������ � ������ ����������� (�.�., �����-�������, ������-������, ������-���� � �.�.)
            if ((DRN == 3 || DRN == 4 || DRN == 5 || DRN == 6) && (direction == 1 || direction == 2 || direction == 3 || direction == 4))
            {
                int share = FLAMEGPU->random.uniform<int>(1, 3);
                x2 = FLAMEGPU->random.uniform(x_min2, x_max2);
                y2 = FLAMEGPU->random.uniform(y_min2, y_max2);
                x3 = FLAMEGPU->random.uniform(x_min3, x_max3);
                y3 = FLAMEGPU->random.uniform(y_min3, y_max3);

                if (share == 1)
                {
                    x = x;
                    y = y;
                }
                if (share == 2)
                {
                    x = x2;
                    y = y2;
                }
                if (share == 3)
                {
                    x = x3;
                    y = y3;
                }
            }



            float vv = FLAMEGPU->environment.getProperty<float>("velocity_of_MGVs");
            v = FLAMEGPU->random.logNormal<float>(log(vv), 0.1); // ��������� �������� ���

            flamegpu::HostNewAgentAPI instance = agent.newAgent();
            instance.setVariable<int>("id", getNextID());
            instance.setVariable<float>("x", x);
            instance.setVariable<float>("y", y);
            instance.setVariable<float>("Ra", PERSONAL_RADIUS);
            instance.setVariable<int>("agent_class", 2);
            instance.setVariable<int>("agent_state", 2);
            instance.setVariable<int>("agent_type", direction);
            instance.setVariable<float>("velocity", v);
            instance.setVariable<float>("rotation", 0);
            instance.setVariable<float>("neighbour_distance", 10000);
            instance.setVariable<float>("other_neighbour_distance", 10000);
            n++;
        }
    }


}

FLAMEGPU_EXIT_CONDITION(exit_condition) {

    if (FLAMEGPU->getStepCounter() >= TIME_STOP - 1)
        return  flamegpu::EXIT;  // ��������� ���������
    else
        return  flamegpu::CONTINUE;  // ����������� ���������
}


FLAMEGPU_HOST_FUNCTION(agents_data_updating) {

    std::lock_guard<std::mutex> lock(m);
    
    std::vector<Points_ugvs> agent_xy;
    agent_xy.clear();

    flamegpu::HostAgentAPI agent = FLAMEGPU->agent("agent-vehicles");
    
    flamegpu::DeviceAgentVector population1 = agent.getPopulationData();
    
    auto traffic = FLAMEGPU->environment.getMacroProperty<uint32_t>("Traffic");

    unsigned int accidents_count = 0;
    unsigned int traffic_count = traffic;

    for (int i = 0; i < agent.count(); i++)
    {
        flamegpu::AgentVector::Agent instance = population1[i];

        x_a[i] = instance.getVariable<float>("x");
        y_a[i] = instance.getVariable<float>("y");
        r_a[i] = instance.getVariable<float>("Ra");
        rot_a[i] = instance.getVariable<float>("rotation");
        agent_class[i] = instance.getVariable<int>("agent_class");
        agent_state[i] = instance.getVariable<int>("agent_state");

        //if (agent_class[i] == 1 && agent_state[i] != 1)
            agent_xy.push_back(Points_ugvs(x_a[i], y_a[i]));


        if (agent_state[i] == 1)
            accidents_count++;
    }


    objective = accidents_count;

    printf("%u, %u \n", traffic_count, accidents_count);

    a_size = agent.count();
   
    
     //���������� ��������� ������������� ������������� ********************************************************************** 
    
     // clustering call
    int npoints = agent_xy.size();
    if (npoints > 2)
    {

        double* distmat = new double[(npoints * (npoints - 1)) / 2];
        int k, i, j;
        for (i = k = 0; i < npoints; i++) {
            for (j = i + 1; j < npoints; j++) {
                // compute distance between agents 
                distmat[k] = distance(agent_xy[i], agent_xy[j]);
                k++;
            }
        }

        int* merge = new int[2 * (npoints - 1)];
        double* height = new double[npoints - 1];
        hclust_fast(npoints, distmat, HCLUST_METHOD_MEDIAN, merge, height);

        int* labels = new int[npoints];
        //cutree_k(npoints, merge, 2, labels); // 2 - ��������
        cutree_cdist(npoints, merge, height, 2 * PERSONAL_RADIUS, labels); // ����������� ���������� ����� ���������� - ��������� ������ ������� ������������

        std::vector <int> cluster_id;
        cluster_id.clear();
    
        int f = 0;
        for (int i = 0; i < npoints; i++) {
            f = 0;
            for (int j = 0; j < cluster_id.size(); j++) {
                if (labels[i] == cluster_id[j])
                {
                    f = 1; // ������� ������
                    break;
                }
            }
            if (f == 0)
                cluster_id.push_back(labels[i]);
        }

        //���������� ������� ���������, � ������� ����� ���������� ��� � ���������� ���������� � ������-��������
        unsigned int counter = 0;
        std::vector <CLUSTERS_UGVs> ugvs;

        ugvs.clear();
        jam_size = 0;
        jam_count = 0;
        for (int i = 0; i < cluster_id.size(); i++)
        {
            int r = 0;
            double x_cluster = 0;
            double y_cluster = 0;
            int d_cluster = 0;


            for (int j = 0; j < npoints; j++)
            {
                if (cluster_id[i] == labels[j])
                {
                    //���������� ������� ���������
                    x_cluster = x_cluster + agent_xy[j].x;
                    y_cluster = y_cluster + agent_xy[j].y;
                    d_cluster++; // ���������� ������� � ��������
                    r++;
                }

            }

            ugvs.push_back(CLUSTERS_UGVs((double)x_cluster / r, (double)y_cluster / r, d_cluster));

            counter++;
        }

        // ���������� ��������� �� ���������� ��� � ��������
        std::sort(ugvs.begin(), ugvs.end(), my_clusters_comparison);
        
        for (int i = 0; i < ugvs.size(); i++)
        {
            if (ugvs[i].d_c > 3) // ���� �������� ����� �� ����� �� � ��������, �� ��� ������
            {
                x_jam[jam_count] = ugvs[i].x_c;
                y_jam[jam_count] = ugvs[i].y_c;
                density_jam[jam_count] = ugvs[i].d_c; // ��������� �������� �������
                jam_size+= ugvs[i].d_c; // ����� ���������� �������� �������
                jam_count++;
            }
        }

        float avg_jams_density = 0;
        if (jam_count > 0)
        {
            avg_jams_density = (float)jam_size / jam_count;
            printf("TRAFFIC JAMS: %f\n", avg_jams_density);
        }

        if (out.is_open() && VIS_MODE == 1) // �������� ����������� ��� ���������� �������
        {
            out << avg_jams_density << std::endl;
        }
        if (FLAMEGPU->getStepCounter() == TIME_STOP - 1 && VIS_MODE == 1)
            out.close();

        // clean up
        delete[] distmat;
        delete[] merge;
        delete[] height;
        delete[] labels;

    

    }
  
    population1.syncChanges();
   //  population1.purgeCache();
   //population1.purgeCache();
}


FLAMEGPU_AGENT_FUNCTION(agent_move, flamegpu::MessageNone, flamegpu::MessageNone) {
    // Behaviour goes here

    int direction = FLAMEGPU->getVariable<int>("agent_type");
    FLAMEGPU->setVariable<float>("rotation", 0);

    int x_min = FLAMEGPU->environment.getProperty<unsigned int>("indent_x");
    int y_min = FLAMEGPU->environment.getProperty<unsigned int>("indent_y");

    int R1 = FLAMEGPU->environment.getProperty<unsigned int>("R1");
    int L = FLAMEGPU->environment.getProperty<unsigned int>("L");
    int w = FLAMEGPU->environment.getProperty<unsigned int>("w");
    int R2 = R1 + w / 2;
    int R3 = R2 + w / 2;
    int R4 = R3 + w / 2;
    int R5 = R4 + w / 2;
    int sign = -1;
    int c1 = 100;

    float x_center = x_min + L / 2;
    float y_center = y_min + L / 2;

    double distance = distance_agents(FLAMEGPU->getVariable<float>("x"), x_center, FLAMEGPU->getVariable<float>("y"), y_center);

    double alpha = -0.1 * FLAMEGPU->getVariable<float>("velocity") / distance;
    double beta = atan2((FLAMEGPU->getVariable<float>("y") - y_center), (FLAMEGPU->getVariable<float>("x") - x_center));

    double prob = FLAMEGPU->random.uniform<float>();

    //����������� ������������ �������
    int DRN = FLAMEGPU->environment.getProperty<unsigned int>("DRN");
    if (DRN == 1)
        { 
        if (distance > R1 && distance < R3 && prob < 0.1 && FLAMEGPU->getVariable<int>("agent_state") != 1)
            FLAMEGPU->setVariable<int>("agent_state", 3); // �������� ��������
        }

    if (DRN == 2 || DRN == 5 ||  DRN == 6)
        {
        if(  ((distance > R1 && distance < R3 && prob < 0.05 && FLAMEGPU->getVariable<int>("agent_state") != 1) ||
             (distance > R3 && distance < R5 && prob < 0.05 && FLAMEGPU->getVariable<int>("agent_state") != 1) ) &&
			  FLAMEGPU->getVariable<int>("agent_state")==2)
             
            FLAMEGPU->setVariable<int>("agent_state", 3); // �������� ��������
        }


       //���� ����� ��������� � ���������� ���������
       if(FLAMEGPU->getVariable<int>("agent_state")!=1)
       { 
           
           if (FLAMEGPU->random.uniform<float>() > 0.5)
               sign = 1;

           //���������� ��������� ��
           if (FLAMEGPU->getVariable<float>("neighbour_distance") > 2 * FLAMEGPU->getVariable<float>("threshold_distance") &&
               FLAMEGPU->getVariable<float>("neighbour_angle") < 5) // ��� ����������� �������
           {
               float vv = 0.0;
               if (FLAMEGPU->getVariable<int>("agent_class") == 1)
                   vv = FLAMEGPU->environment.getProperty<float>("velocity_of_UGVs");
               if (FLAMEGPU->getVariable<int>("agent_class") == 2)
                   vv = FLAMEGPU->environment.getProperty<float>("velocity_of_MGVs");

               FLAMEGPU->setVariable<float>("velocity", FLAMEGPU->random.logNormal<float>(log(vv), 0.1));

           }

           if (FLAMEGPU->getVariable<float>("neighbour_distance") <= 2 * FLAMEGPU->getVariable<float>("threshold_distance") &&
               FLAMEGPU->getVariable<float>("neighbour_angle") < 5) // ���� ����������� ������� �� ��
           {
               FLAMEGPU->setVariable<float>("velocity", 0.01 * FLAMEGPU->getVariable<float>("velocity"));
               //printf("Deceleration!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!");
           }

           if (DRN == 1) // ������ ��� ����� ���
           { 
                switch (direction) {
          
            case 1: //�������� �����-�������

                if (FLAMEGPU->getVariable<int>("agent_state")==3 && 
                    FLAMEGPU->getVariable<float>("x") >= x_min + L/2 &&
					FLAMEGPU->getVariable<float>("y") >= y_min + L/2 - w &&
                    FLAMEGPU->getVariable<float>("y") <= y_min + L/2)
                    FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                if( (FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4) )
                {
                    
                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                        FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity"));

                   
                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") *sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                                 (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 - w &&
                             FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                                 (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2)
                    {
                        
                        FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                            FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                        FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                            FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));
                         
                    }
                  
                    double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        y_potentional2 >= y_min + L/2 - w &&
                        y_potentional2 <= y_min + L/2) 
                    {
                        FLAMEGPU->setVariable<float>("x", x_potentional2);
                        FLAMEGPU->setVariable<float>("y", y_potentional2);
                    }
                    

                }

                if(FLAMEGPU->getVariable<int>("agent_state") == 3)
                    {
                    alpha = 1 * FLAMEGPU->getVariable<float>("velocity") / distance;

                    double x_potential1 = FLAMEGPU->getVariable<float>("x") +  cos(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                                                                (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                    double y_potential1 = FLAMEGPU->getVariable<float>("y") + sin(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                                                                (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));

                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                    {
                    FLAMEGPU->setVariable<float>("x", x_center + distance * cos(alpha + beta) );
                    FLAMEGPU->setVariable<float>("y", y_center + distance * sin(alpha + beta) );
                    }
                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if(pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) >= R1 &&
                            pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential1);
                        FLAMEGPU->setVariable<float>("y", y_potential1);
                    }

                    double x_potential2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potential2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));
                    
                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ���� ��������� �������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential2);
                        FLAMEGPU->setVariable<float>("y", y_potential2);
                    }


                    }

                break;
            
            case 2: // �������� ������-������
                if (FLAMEGPU->getVariable<int>("agent_state") == 3 &&
                    FLAMEGPU->getVariable<float>("x") <= x_min + L/2 &&
					FLAMEGPU->getVariable<float>("y") >= y_min + L / 2  &&
                    FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 + w)
                    FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                {

                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                        FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") - FLAMEGPU->getVariable<float>("velocity"));


                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 &&

                        FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 + w)
                    {

                        FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                            FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                        FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                            FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                    }

                    double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        y_potentional2 >= y_min + L / 2 &&
                        y_potentional2 <= y_min + L / 2 + w)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potentional2);
                        FLAMEGPU->setVariable<float>("y", y_potentional2);
                    }


                }

                if (FLAMEGPU->getVariable<int>("agent_state") == 3)
                {
                    alpha = 1 * FLAMEGPU->getVariable<float>("velocity") / distance;

                    double x_potential1 = FLAMEGPU->getVariable<float>("x") + cos(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                    double y_potential1 = FLAMEGPU->getVariable<float>("y") + sin(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));

                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                    {
                        FLAMEGPU->setVariable<float>("x", x_center + distance * cos(alpha + beta));
                        FLAMEGPU->setVariable<float>("y", y_center + distance * sin(alpha + beta));
                    }
                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if (pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential1);
                        FLAMEGPU->setVariable<float>("y", y_potential1);
                    }

                    double x_potential2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potential2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ���� ��������� �������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential2);
                        FLAMEGPU->setVariable<float>("y", y_potential2);
                    }
                }

               break;
            
            case 3: // �������� �����-�����
                if (FLAMEGPU->getVariable<int>("agent_state") == 3 &&
					FLAMEGPU->getVariable<float>("x") >= x_min + L / 2  &&
                    FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + w &&
                    FLAMEGPU->getVariable<float>("y") >= y_min + L / 2)
                    FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                {

                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                        FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity"));


                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 &&
                             FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2 + w)
                    {

                        FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                            FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                        FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                            FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                    }

                    double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        x_potentional2 >= x_min + L / 2 &&
                        x_potentional2 <= x_min + L / 2 + w)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potentional2);
                        FLAMEGPU->setVariable<float>("y", y_potentional2);
                    }


                }

                if (FLAMEGPU->getVariable<int>("agent_state") == 3)
                {
                    alpha = 1 * FLAMEGPU->getVariable<float>("velocity") / distance;

                    double x_potential1 = FLAMEGPU->getVariable<float>("x") + cos(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                    double y_potential1 = FLAMEGPU->getVariable<float>("y") + sin(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));

                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                    {
                        FLAMEGPU->setVariable<float>("x", x_center + distance * cos(alpha + beta));
                        FLAMEGPU->setVariable<float>("y", y_center + distance * sin(alpha + beta));
                    }
                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if (pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential1);
                        FLAMEGPU->setVariable<float>("y", y_potential1);
                    }

                    double x_potential2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potential2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ���� ��������� �������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential2);
                        FLAMEGPU->setVariable<float>("y", y_potential2);
                    }
                }

                break;
            case 4: // �������� ������-����
                if (FLAMEGPU->getVariable<int>("agent_state") == 3 &&
					FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 - w &&
                    FLAMEGPU->getVariable<float>("x") <= x_min + L / 2  &&
                    FLAMEGPU->getVariable<float>("y") <= y_min + L / 2)
                    FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                {

                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                        FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") - FLAMEGPU->getVariable<float>("velocity"));


                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 - w &&
                        FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2)
                    {

                        FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                            FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                        FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                            FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                            (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                    }

                    double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        x_potentional2 >= x_min + L / 2 - w &&
                        x_potentional2 <= x_min + L / 2)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potentional2);
                        FLAMEGPU->setVariable<float>("y", y_potentional2);
                    }


                }

                if (FLAMEGPU->getVariable<int>("agent_state") == 3)
                {
                    alpha = 1 * FLAMEGPU->getVariable<float>("velocity") / distance;

                    double x_potential1 = FLAMEGPU->getVariable<float>("x") + cos(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                    double y_potential1 = FLAMEGPU->getVariable<float>("y") + sin(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                        (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));

                    if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                    {
                        FLAMEGPU->setVariable<float>("x", x_center + distance * cos(alpha + beta));
                        FLAMEGPU->setVariable<float>("y", y_center + distance * sin(alpha + beta));
                    }
                    // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                    else if (pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential1);
                        FLAMEGPU->setVariable<float>("y", y_potential1);
                    }

                    double x_potential2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                    double y_potential2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                    // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ���� ��������� �������
                    if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) <= R3)
                    {
                        FLAMEGPU->setVariable<float>("x", x_potential2);
                        FLAMEGPU->setVariable<float>("y", y_potential2);
                    }
                }
                break;
            }
           }

           if (DRN == 2 || DRN == 5 || DRN == 4 || DRN == 6) 
           {
               switch (direction) {

               case 1: //�������� �����-�������

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN != 5 || DRN != 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 - w &&
                       FLAMEGPU->getVariable<float>("y") <= y_min + L / 2)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   double prob1 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 - 3 * w &&
                       FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 - 2 * w && prob1 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   double prob2 = FLAMEGPU->random.uniform<float>();

                       if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                           FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 &&
                           FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 - w &&
                           FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 && prob2 < 0.1)
                           FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   double prob3 = FLAMEGPU->random.uniform<float>();

                       if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                           FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 &&
                           FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 + w &&
                           FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 + 2 * w && prob3 < 0.1)
                           FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity"));


                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       else if (
                           
                           (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 - w&&
                           FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 ) ||

                           (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 - 3 * w &&
                               FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 - 2 * w) ||

                           (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 +  w &&
                               FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 + 2 * w)

                               )
                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if  (
                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                           y_potentional2 >= y_min + L / 2 - w &&
                           y_potentional2 <= y_min + L / 2) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               y_potentional2 >= y_min + L / 2 - 3* w &&
                               y_potentional2 <= y_min + L / 2 - 2* w) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               y_potentional2 >= y_min + L / 2 + w &&
                               y_potentional2 <= y_min + L / 2 + 2*w) 
                           )

                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   

                   break;

               case 2: // �������� ������-������
                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN!=5 || DRN!=6) &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 + w)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   prob1 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN != 5 || DRN != 6) &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 - 2 *w &&
                       FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 - w && prob1< 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����


                   prob2 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN != 5 || DRN != 6) &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2  &&
                       FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 + w && prob2 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   prob3 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN != 5 || DRN != 6) &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 + 2*w &&
                       FLAMEGPU->getVariable<float>("y") <= y_min + L / 2 + 3*w && prob3 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����
                   

                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") - FLAMEGPU->getVariable<float>("velocity"));


                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       else if (
                           
                           (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 &&
                           FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 + w) ||

                           (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 - 2 *w &&
                               FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 - w) ||

                           (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 + 2 * w &&
                               FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 + 3 * w)
                               
                               )
                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if  (
                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                           y_potentional2 >= y_min + L / 2 &&
                           y_potentional2 <= y_min + L / 2 + w) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               y_potentional2 >= y_min + L / 2 - 2 *w &&
                               y_potentional2 <= y_min + L / 2 - w) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               y_potentional2 >= y_min + L / 2 + 2 * w &&
                               y_potentional2 <= y_min + L / 2 + 3 * w)
                           )

                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   break;

               case 3: // �������� �����-�����
                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN!=5 || DRN!=6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2  &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + w &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   prob1 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 - 2*w &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 - w &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 && prob1 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����


                   prob2 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2  &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + w &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 && prob2 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   prob3 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 + 2*w &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + 3*w &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 && prob3 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����


                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity"));


                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       else if (
                           (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 &&
                           FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2 + w) ||

                           (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 - 2 * w &&
                               FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2 - w) ||

                           (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 + 2 * w &&
                               FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2 + 3 * w) 
                              )

                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if (
                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                           x_potentional2 >= x_min + L / 2 &&
                           x_potentional2 <= x_min + L / 2 + w) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               x_potentional2 >= x_min + L / 2 - 2 * w &&
                               x_potentional2 <= x_min + L / 2 -  w) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               x_potentional2 >= x_min + L / 2 + 2 * w &&
                               x_potentional2 <= x_min + L / 2 + 3 * w)
                           )
                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   break;

               case 4: // �������� ������-����
                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN != 5 || DRN != 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 - w &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") <= y_min + L / 2)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   prob1 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 - 3 * w &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 - 2 * w &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 && prob1 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����


                   prob2 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 - 2&&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 && prob2 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   prob3 = FLAMEGPU->random.uniform<float>();

                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 && (DRN == 5 || DRN == 6) &&
                       FLAMEGPU->getVariable<float>("x") >= x_min + L / 2 + w &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + 2 * w &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2 && prob3 < 0.1)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") - FLAMEGPU->getVariable<float>("velocity"));


                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       else if (
                           (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 - w &&
                           FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2) ||

                           (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 - 3 * w &&
                           FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2 - 2 * w) ||

                           (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 + w &&
                           FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2 + 2 * w)
                               )

                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if  (
                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                           x_potentional2 >= x_min + L / 2 - w &&
                           x_potentional2 <= x_min + L / 2) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               x_potentional2 >= x_min + L / 2 - 3 * w &&
                               x_potentional2 <= x_min + L / 2 - 2 * w) ||

                           (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                               x_potentional2 >= x_min + L / 2 + w &&
                               x_potentional2 <= x_min + L / 2 + 2 * w)
                           )
                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   break;

               case 5: // �������� �-�
                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + w / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                       {
                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity")*cos(45 * M_PI / 180));
                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity")*sin(45 * M_PI / 180));
                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") +
                           FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") +
                           FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));


                       double x2 = x_min + L - (double)w / sin(45 * M_PI / 180);
                       double y2 = y_min + L;
                       double x1 = x_min - (double)w / sin(45 * M_PI / 180);
                       double y1 = y_min;


                       double x4 = x_min + L;
                       double y4 = y_min + L;
                       double x3 = x_min;
                       double y3 = y_min;

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       double sign_road1 =  (x2 - x1)*(y_potentional2 - y1) - (y2-y1)*(x_potentional2 - x1);
                       double sign_road2 =  (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);
                       

                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("neighbour_distance") <= FLAMEGPU->getVariable<float>("threshold_distance") &  // ���� ����������
                           sign_road1 < 0 && sign_road2 > 0)
                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       sign_road1 = (x2 - x1) * (y_potentional2 - y1) - (y2 - y1) * (x_potentional2 - x1);
                       sign_road2 = (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);

                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &
                           sign_road1 < 0 && sign_road2 > 0 )
                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   break;

               case 6: // �������� �-�
                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + w / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                       {
                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") - FLAMEGPU->getVariable<float>("velocity") * cos(45 * M_PI / 180));
                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") - FLAMEGPU->getVariable<float>("velocity") * sin(45 * M_PI / 180));
                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") -
                           FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) -
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") -
                           FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) -
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));


                       double x2 = x_min + L;
                       double y2 = y_min + L;
                       double x1 = x_min;
                       double y1 = y_min;

                       double x4 = x_min + L + (double)w / sin(45 * M_PI / 180);
                       double y4 = y_min + L;
                       double x3 = x_min + (double)w / sin(45 * M_PI / 180);
                       double y3 = y_min;

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       double sign_road1 = (x2 - x1) * (y_potentional2 - y1) - (y2 - y1) * (x_potentional2 - x1);
                       double sign_road2 = (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);


                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("neighbour_distance") <= FLAMEGPU->getVariable<float>("threshold_distance") &  // ���� �����������
                           sign_road1 < 0 && sign_road2 > 0)
                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") -
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) -
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") -
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) -
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       x_potentional2 = FLAMEGPU->getVariable<float>("x") - (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       y_potentional2 = FLAMEGPU->getVariable<float>("y") - (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       sign_road1 = (x2 - x1) * (y_potentional2 - y1) - (y2 - y1) * (x_potentional2 - x1);
                       sign_road2 = (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);

                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &
                           sign_road1 < 0 && sign_road2 > 0)
                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   break;

               case 7: // �������� �-�
                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + w / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                       {
                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(-45 * M_PI / 180));
                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(-45 * M_PI / 180));
                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") +
                           FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") +
                           FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));


                       double x2 = x_min + L;
                       double y2 = y_min;
                       double x1 = x_min;
                       double y1 = y_min + L;

                       double x4 = x_min + L - (double)w / sin(45 * M_PI / 180);
                       double y4 = y_min;
                       double x3 = x_min - (double)w / sin(45 * M_PI / 180);
                       double y3 = y_min + L;

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       double sign_road1 = (x2 - x1) * (y_potentional2 - y1) - (y2 - y1) * (x_potentional2 - x1);
                       double sign_road2 = (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);


                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("neighbour_distance") <= FLAMEGPU->getVariable<float>("threshold_distance") &  // ���� ����������
                           sign_road1 < 0 && sign_road2 > 0)
                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       sign_road1 = (x2 - x1) * (y_potentional2 - y1) - (y2 - y1) * (x_potentional2 - x1);
                       sign_road2 = (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);

                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &
                           sign_road1 < 0 && sign_road2 > 0)
                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   break;

               case 8: // �������� �-�
                   if (FLAMEGPU->getVariable<int>("agent_state") == 3 &&
                       FLAMEGPU->getVariable<float>("x") <= x_min + L / 2 + w / 2 &&
                       FLAMEGPU->getVariable<float>("y") >= y_min + L / 2)
                       FLAMEGPU->setVariable<int>("agent_state", 4); // ����� � �����

                   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
                   {

                       if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
                       {
                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") - FLAMEGPU->getVariable<float>("velocity") * cos(45 * M_PI / 180));
                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(45 * M_PI / 180));
                       }

                       double x_potentional2 = FLAMEGPU->getVariable<float>("x") -
                           FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) -
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                       double y_potentional2 = FLAMEGPU->getVariable<float>("y") +
                           FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                           (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));


                       double x2 = x_min + L + (double)w / sin(45 * M_PI / 180);
                       double y2 = y_min;
                       double x1 = x_min  + (double)w / sin(45 * M_PI / 180);
                       double y1 = y_min + L;

                       double x4 = x_min + L;
                       double y4 = y_min;
                       double x3 = x_min;
                       double y3 = y_min + L;

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       double sign_road1 = (x2 - x1) * (y_potentional2 - y1) - (y2 - y1) * (x_potentional2 - x1);
                       double sign_road2 = (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);


                       // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("neighbour_distance") <= FLAMEGPU->getVariable<float>("threshold_distance") &  // ���� ����������
                           sign_road1 < 0 && sign_road2 > 0)
                       {

                           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") -
                               FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) -
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

                           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
                               FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
                               (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

                       }

                       x_potentional2 = FLAMEGPU->getVariable<float>("x") - (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                       y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                       // ���� sign_road > 0, �� ���� ������, ��� sign_road < 0, �� ���� ������������ ������ 
                       sign_road1 = (x2 - x1) * (y_potentional2 - y1) - (y2 - y1) * (x_potentional2 - x1);
                       sign_road2 = (x4 - x3) * (y_potentional2 - y3) - (y4 - y3) * (x_potentional2 - x3);

                       // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
                       if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &
                           sign_road1 < 0 && sign_road2 > 0)
                       {
                           FLAMEGPU->setVariable<float>("x", x_potentional2);
                           FLAMEGPU->setVariable<float>("y", y_potentional2);
                       }


                   }

                   break;
               }

               if (FLAMEGPU->getVariable<int>("agent_state") == 3)
               {
                   alpha =   1 * FLAMEGPU->getVariable<float>("velocity") / distance;

                  
                   if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance") &&
                          (pow(pow(distance * cos(-alpha + beta), 2) + pow(distance * sin(-alpha + beta), 2), 0.5) >= R1 &&
                           pow(pow(distance * cos(-alpha + beta), 2) + pow(distance * sin(-alpha + beta), 2), 0.5) <= R3)) // ��� �����������
                   {
                       FLAMEGPU->setVariable<float>("x", x_center + distance * cos(-alpha + beta));
                       FLAMEGPU->setVariable<float>("y", y_center + distance * sin(-alpha + beta));
                   }

                   if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance") &&
                       (pow(pow(distance * cos(alpha + beta), 2) + pow(distance * sin(alpha + beta), 2), 0.5) >= R3 &&
                        pow(pow(distance * cos(alpha + beta), 2) + pow(distance * sin(alpha + beta), 2), 0.5) <= R5)) // ��� �����������
                   {
                       FLAMEGPU->setVariable<float>("x", x_center + distance * cos(alpha + beta));
                       FLAMEGPU->setVariable<float>("y", y_center + distance * sin(alpha + beta));
                   }

                   double x_potential1 = FLAMEGPU->getVariable<float>("x") + cos(-alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                       (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                   double y_potential1 = FLAMEGPU->getVariable<float>("y") + sin(-alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) +
                       (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));

                   // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
                   if (FLAMEGPU->getVariable<float>("neighbour_distance") <= FLAMEGPU->getVariable<float>("threshold_distance") &&
                       (pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) >= R1 &&
                        pow(pow(x_potential1 - x_center, 2) + pow(y_potential1 - y_center, 2), 0.5) <= R3))
                   {
                       FLAMEGPU->setVariable<float>("x", x_potential1);
                       FLAMEGPU->setVariable<float>("y", y_potential1);
                   }

                   double x_potential2 = FLAMEGPU->getVariable<float>("x") + cos(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) -
                       (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma"));

                   double y_potential2 = FLAMEGPU->getVariable<float>("y") + sin(alpha + beta + sign * FLAMEGPU->getVariable<float>("omega")) -
                       (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma"));

                   if (FLAMEGPU->getVariable<float>("neighbour_distance") <= FLAMEGPU->getVariable<float>("threshold_distance") &&
                       (pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) >= R3 &&
                       pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) <= R5))

                   {
                       FLAMEGPU->setVariable<float>("x", x_potential2);
                       FLAMEGPU->setVariable<float>("y", y_potential2);
                   }

                   x_potential2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                   y_potential2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                   // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ���� ��������� �������
                   if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                       (pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) >= R1 &&
                           pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) <= R3))

                   {
                       FLAMEGPU->setVariable<float>("x", x_potential2);
                       FLAMEGPU->setVariable<float>("y", y_potential2);
                   }

                   x_potential2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
                   y_potential2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));

                   // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ���� ��������� �������
                   if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
                       (pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) >= R3 &&
                           pow(pow(x_potential2 - x_center, 2) + pow(y_potential2 - y_center, 2), 0.5) <= R5))
                   {
                       FLAMEGPU->setVariable<float>("x", x_potential2);
                       FLAMEGPU->setVariable<float>("y", y_potential2);
                   }
               }
           }

		   if (DRN == 3) // ������ ���
		   {
			   switch (direction) {

			   case 1: //�������� �����-�������

				   
				   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
				   {

					   if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
						   FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity"));


					   // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
					   else if (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 &&
						   FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 + w)
					   {

						   FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
							   FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

						   FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
							   FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

					   }

					   double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
					   double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


					   // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
					   if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
						   y_potentional2 >= y_min + L / 2 &&
						   y_potentional2 <= y_min + L / 2)
					   {
						   FLAMEGPU->setVariable<float>("x", x_potentional2);
						   FLAMEGPU->setVariable<float>("y", y_potentional2);
					   }


				   }

				   break;

			   case 2: // �������� ������-������
				
				   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
				   {

					   if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
						   FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") - FLAMEGPU->getVariable<float>("velocity"));


					   // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
					   else if (FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) >= y_min + L / 2 &&

						   FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")) <= y_min + L / 2 + w)
					   {

						   FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
							   FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

						   FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
							   FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

					   }

					   double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
					   double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


					   // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
					   if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
						   y_potentional2 >= y_min + L / 2 &&
						   y_potentional2 <= y_min + L / 2)
					   {
						   FLAMEGPU->setVariable<float>("x", x_potentional2);
						   FLAMEGPU->setVariable<float>("y", y_potentional2);
					   }


				   }

				   break;

			   case 3: // �������� �����-�����
				   
				   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
				   {

					   if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
						   FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") + FLAMEGPU->getVariable<float>("velocity"));


					   // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
					   else if (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 &&
						   FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2 + w)
					   {

						   FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
							   FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

						   FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
							   FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

					   }

					   double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
					   double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


					   // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
					   if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
						   x_potentional2 >= x_min + L / 2 &&
						   x_potentional2 <= x_min + L / 2 + w)
					   {
						   FLAMEGPU->setVariable<float>("x", x_potentional2);
						   FLAMEGPU->setVariable<float>("y", y_potentional2);
					   }


				   }

				   break;
			   case 4: // �������� ������-����
				  
				   if ((FLAMEGPU->getVariable<int>("agent_state") == 2 || FLAMEGPU->getVariable<int>("agent_state") == 4))
				   {

					   if (FLAMEGPU->getVariable<float>("neighbour_distance") > FLAMEGPU->getVariable<float>("threshold_distance")) // ��� �����������
						   FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") - FLAMEGPU->getVariable<float>("velocity"));


					   // ����� ����������� ������� ���������� ������-�� ��� ������� ������� �� ��������� ������
					   else if (FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) >= x_min + L / 2 - w &&
						   FLAMEGPU->getVariable<float>("x") + FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
						   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")) <= x_min + L / 2)
					   {

						   FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") +
							   FLAMEGPU->getVariable<float>("velocity") * cos(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("gamma")));

						   FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y") +
							   FLAMEGPU->getVariable<float>("velocity") * sin(sign * FLAMEGPU->getVariable<float>("omega")) +
							   (c1 / FLAMEGPU->getVariable<float>("neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("gamma")));

					   }

					   double x_potentional2 = FLAMEGPU->getVariable<float>("x") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * cos(FLAMEGPU->getVariable<float>("other_gamma"));
					   double y_potentional2 = FLAMEGPU->getVariable<float>("y") + (c1 / FLAMEGPU->getVariable<float>("other_neighbour_distance")) * sin(FLAMEGPU->getVariable<float>("other_gamma"));


					   // �������� � ��������������� �� ���������� ������-�� �����������  ��� ������� ������� �� ��������� ������
					   if (FLAMEGPU->getVariable<float>("other_neighbour_distance") < FLAMEGPU->getVariable<float>("other_threshold_distance") &&
						   x_potentional2 >= x_min + L / 2 - w &&
						   x_potentional2 <= x_min + L / 2)
					   {
						   FLAMEGPU->setVariable<float>("x", x_potentional2);
						   FLAMEGPU->setVariable<float>("y", y_potentional2);
					   }


				   }

				   break;
			   }
		   }
		   
       }

       if (FLAMEGPU->getVariable<int>("agent_state") == 1)
       {
           FLAMEGPU->setVariable<float>("x", FLAMEGPU->getVariable<float>("x") );
           FLAMEGPU->setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
       }

        
        double rot = atan2((FLAMEGPU->getVariable<float>("y") - FLAMEGPU->getVariable<float>("y_past")), (FLAMEGPU->getVariable<float>("x") - FLAMEGPU->getVariable<float>("x_past")));
        if(FLAMEGPU->getVariable<int>("agent_state") != 1)
          FLAMEGPU->setVariable<float>("rotation", rot);
        else
          FLAMEGPU->setVariable<float>("rotation", FLAMEGPU->getVariable<float>("rotation_past") );

        //���������� �������� ��������� ������-��
        FLAMEGPU->setVariable<float>("x_past", FLAMEGPU->getVariable<float>("x") );
        FLAMEGPU->setVariable<float>("y_past", FLAMEGPU->getVariable<float>("y") );
        FLAMEGPU->setVariable<float>("rotation_past", FLAMEGPU->getVariable<float>("rotation"));


    return flamegpu::ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(update_agent_state, flamegpu::MessageNone, flamegpu::MessageSpatial2D)
{
    FLAMEGPU->setVariable<float>("threshold_distance", 0);
    FLAMEGPU->setVariable<float>("other_threshold_distance", 0);

    int agent_type = FLAMEGPU->getVariable<int>("agent_type");
    int agent_state = FLAMEGPU->getVariable<int>("agent_state");

    float x = FLAMEGPU->getVariable<float>("x");
    float y = FLAMEGPU->getVariable<float>("y");

    int x_min = FLAMEGPU->environment.getProperty<unsigned int>("indent_x");
    int x_max = FLAMEGPU->environment.getProperty<unsigned int>("L") + FLAMEGPU->environment.getProperty<unsigned int>("indent_x");
    int y_min = FLAMEGPU->environment.getProperty<unsigned int>("indent_y");
    int y_max = FLAMEGPU->environment.getProperty<unsigned int>("L") + FLAMEGPU->environment.getProperty<unsigned int>("indent_y");

    auto traffic = FLAMEGPU->environment.getMacroProperty<uint32_t>("Traffic");

    //������� �������-�� �� ���
    if (agent_type == 1 && x > x_max || agent_type == 2 && x < x_min || agent_type == 3 && y > y_max || agent_type == 4 && y < y_min ||
        agent_type == 5 && y > y_max || agent_type == 6 && y < y_min || agent_type == 8 && y > y_max || agent_type == 7 && y < y_min)
        {
        unsigned int vehicles_count = traffic++;
        //printf("%u \n", vehicles_count); // ������� ��������� ������
        return flamegpu::DEAD;
        }
    else
    {
        //�������� ������ � �������������� ������-�� � ��� ����������
        FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
        FLAMEGPU->message_out.setVariable<float>("Ra", FLAMEGPU->getVariable<float>("Ra"));
        FLAMEGPU->message_out.setVariable<int>("agent_type", FLAMEGPU->getVariable<int>("agent_type"));
        FLAMEGPU->message_out.setVariable<int>("agent_state", FLAMEGPU->getVariable<int>("agent_state"));
        
        FLAMEGPU->message_out.setLocation(
            FLAMEGPU->getVariable<float>("x"),
            FLAMEGPU->getVariable<float>("y"));

        return flamegpu::ALIVE;
    }
}


//������ ��������� ������ ������ ������ � ��������� ������� ������� ������������ ������ � ��� ��������
FLAMEGPU_AGENT_FUNCTION(density_estimation, flamegpu::MessageSpatial2D, flamegpu::MessageNone) 
{
    const float RADIUS = FLAMEGPU->message_in.radius();
    float density = 0.0;
    float separation = 0.0;

    FLAMEGPU->setVariable<float>("neighbour_distance", 10000);
    FLAMEGPU->setVariable<float>("other_neighbour_distance", 10000);

    // Get this agent's x, y, z variables
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    int flag = 0;
    

    // For each message in the message list which was output by a nearby agent
    for (const auto& message : FLAMEGPU->message_in(x1, y1)) {
        const float x2 = message.getVariable<float>("x");
        const float y2 = message.getVariable<float>("y");

        // Calculate the distance to check the message is in range
        float x21 = x2 - x1;
        float y21 = y2 - y1;

        //��������� ���� ����������� (������) �� ��������� ������, � ��������
        float dir_angle = (180/M_PI) * atan2(y21, x21);
        

        //printf("%f\n", (180 / M_PI)*atan2(y21, x21));

        if (message.getVariable<int>("id") != FLAMEGPU->getVariable<int>("id"))
        {
            separation = sqrt(x21 * x21 + y21 * y21);
            if (separation < INTERACTION_RADIUS) {
                // Process the message
                density++;
                //int idFromMessage = message.getVariable<int>("id");
                flag = 1;
            }

            if (separation == 0)
                separation = 1000000;


            if (separation < FLAMEGPU->getVariable<float>("neighbour_distance") && abs(dir_angle) <= 0.001)
            {
                //������ ���������� �� ���������� ������ � ������ ���������� ���������� ������, �������������� ������� �� ���� ��������

                FLAMEGPU->setVariable<float>("neighbour_distance", separation);
                FLAMEGPU->setVariable<float>("threshold_distance", FLAMEGPU->getVariable<float>("Ra") + message.getVariable<float>("Ra")); // ��������� ���������� ����� ��������: ����� �������� ������ ����������� �������� �������
                FLAMEGPU->setVariable<int>("id_neighbour", message.getVariable<int>("id"));
                FLAMEGPU->setVariable<float>("x_neighbour", message.getVariable<float>("x")); // ���������� ���������� ������
                FLAMEGPU->setVariable<float>("y_neighbour", message.getVariable<float>("y"));
                FLAMEGPU->setVariable<float>("omega", (M_PI / 4) +    (atan2(message.getVariable<float>("y") - FLAMEGPU->getVariable<float>("y") + (FLAMEGPU->getVariable<float>("Ra") + message.getVariable<float>("Ra")) * sin(M_PI / 4),
                                                                         message.getVariable<float>("x") - FLAMEGPU->getVariable<float>("x") + (FLAMEGPU->getVariable<float>("Ra") + message.getVariable<float>("Ra")) * cos(M_PI / 4))));
                
               // FLAMEGPU->setVariable<float>("omega", (M_PI / 4));

                FLAMEGPU->setVariable<float>("gamma", M_PI + (atan2(message.getVariable<float>("y") - FLAMEGPU->getVariable<float>("y"),
                                                                    message.getVariable<float>("x") - FLAMEGPU->getVariable<float>("x"))));
                FLAMEGPU->setVariable<float>("neighbour_angle", dir_angle); // ���� ����������� �� ���������� ������

            }

            if (separation < FLAMEGPU->getVariable<float>("other_neighbour_distance") && abs(dir_angle) > 0.001)
            {
                //������ ���������� �� ���������� ������ � ������ ���������� ���������� ������

                FLAMEGPU->setVariable<float>("other_neighbour_distance", separation);
                FLAMEGPU->setVariable<float>("other_threshold_distance", FLAMEGPU->getVariable<float>("Ra") + message.getVariable<float>("Ra")); // ��������� ���������� ����� ��������: ����� �������� ������ ����������� �������� �������
                FLAMEGPU->setVariable<float>("other_gamma", M_PI + (atan2(message.getVariable<float>("y") - FLAMEGPU->getVariable<float>("y"),
                                                                          message.getVariable<float>("x") - FLAMEGPU->getVariable<float>("x"))));
                FLAMEGPU->setVariable<float>("neighbour_angle", dir_angle); // ���� ����������� �� ���������� ������
            }

            int agent_type = FLAMEGPU->getVariable<int>("agent_type");
            int agent_state = FLAMEGPU->getVariable<int>("agent_state");

            float x = FLAMEGPU->getVariable<float>("x");
            float y = FLAMEGPU->getVariable<float>("y");

            int x_min = FLAMEGPU->environment.getProperty<unsigned int>("indent_x");
            int x_max = FLAMEGPU->environment.getProperty<unsigned int>("L") + FLAMEGPU->environment.getProperty<unsigned int>("indent_x");
            int y_min = FLAMEGPU->environment.getProperty<unsigned int>("indent_y");
            int y_max = FLAMEGPU->environment.getProperty<unsigned int>("L") + FLAMEGPU->environment.getProperty<unsigned int>("indent_y");
        
            if (separation < 1 && message.getVariable<int>("agent_state") != 1 &&
                !(agent_type == 1 && x > x_max || agent_type == 2 && x < x_min || agent_type == 3 && y > y_max || agent_type == 4 && y < y_min) &&
                  FLAMEGPU->getVariable<float>("neighbour_distance") < 10000 )
                { 
                FLAMEGPU->setVariable<int>("agent_state", 1); // ��������� ��������

               
                printf("%f, %f, %f, %f, %f  TRAFFIC ACCIDENT \n", separation, 
                                                                  FLAMEGPU->getVariable<float>("neighbour_distance"), 
                                                                  FLAMEGPU->getVariable<float>("threshold_distance"),
                                                                  FLAMEGPU->getVariable<float>("omega"),
                                                                  FLAMEGPU->getVariable<float>("gamma") );
              

                }


        }
    }

   
    //��������� �������� ��������� ������ ������ � �������� ������� ��� ������� ������������

    FLAMEGPU->setVariable<float>("density", density);

    double gamma = 1.0;
    if (FLAMEGPU->getVariable<int>("agent_class") == 2)
        gamma = 4; // ���
    else if (FLAMEGPU->getVariable<int>("agent_class") == 1)
        gamma = 0.8; // ���


    if (FLAMEGPU->getVariable<float>("density") <= 1)
        FLAMEGPU->setVariable<float>("Ra", PERSONAL_RADIUS);
    if (FLAMEGPU->getVariable<float>("density") > 1 && FLAMEGPU->getVariable<float>("density") < 20 && FLAMEGPU->getVariable<int>("agent_state") != 1)
        FLAMEGPU->setVariable<float>("Ra", new_radius(FLAMEGPU->getVariable<float>("density"), 1) );
    if (FLAMEGPU->getVariable<float>("density") >= 20 && FLAMEGPU->getVariable<int>("agent_state") != 1)
        FLAMEGPU->setVariable<float>("Ra", new_radius(FLAMEGPU->getVariable<float>("density"), gamma) );


    //printf("%f\n", FLAMEGPU->getVariable<float>("Ra") );

    return flamegpu::ALIVE;
}


int main(int argc, const char** argv) {
flamegpu::ModelDescription model("Transportation");

const float RADIUS = 150.0f;


flamegpu::MessageSpatial2D::Description& message_distance = model.newMessage<flamegpu::MessageSpatial2D>("location_agent");
{
    message_distance.newVariable<int>("id");
    message_distance.newVariable<float>("Ra");
    message_distance.newVariable<int>("agent_type");
    message_distance.newVariable<int>("agent_state");

    message_distance.setRadius(RADIUS);
    message_distance.setMin(0, 0);
    message_distance.setMax(window_width, window_height);
}

flamegpu::AgentDescription& agent = model.newAgent("agent-vehicles");
{
    agent.newVariable<int>("id"); // ID
    agent.newVariable<float>("x"); // ���������� ������ � ���������� ������� ���������
    agent.newVariable<float>("y");
    agent.newVariable<float>("x_past"); // ��������� �������� ��������� ������-��
    agent.newVariable<float>("y_past");
    agent.newVariable<float>("rotation_past");
    agent.newVariable<int>("agent_class"); // 1 - ���, 2 - ���
    agent.newVariable<int>("agent_state"); // 0 - ����������, 1 - ���������
    agent.newVariable<int>("agent_type"); // 1 - �����-�������, 2-������-������, 3-�����-�����, 4-������-����
    agent.newVariable<float>("Ra"); // ������ ������� ������������ ������
    agent.newVariable<float>("rotation"); // ���� �������� ��
    agent.newVariable<float>("velocity"); // �������� ������
    agent.newVariable<float>("density"); // ��������� ��������� ������ ������ ������
    agent.newVariable<float>("alpha"); // ����, ������������ ����������� �������� ������ (������)
    agent.newVariable<float>("omega"); // ����, ������� ����������� �����������
    agent.newVariable<float>("gamma"); // ����, ������� �� ����������

    agent.newVariable<float>("neighbour_distance"); // ��������� ���������� ������, ������������� ������� �� ���� ��������
    agent.newVariable<float>("threshold_distance");
    agent.newVariable<float>("id_neighbour");
    agent.newVariable<float>("x_neighbour");
    agent.newVariable<float>("y_neighbour");
    agent.newVariable<float>("neighbour_angle");

    agent.newVariable<float>("other_neighbour_distance"); // ��������� ���������� ������
    agent.newVariable<float>("other_threshold_distance");
    agent.newVariable<float>("other_gamma");
        
}

//������� ����������� �������-��
flamegpu::AgentFunctionDescription& agent_fn1_description = agent.newFunction("agent_move", agent_move);

//������� ���������� ��������� �������  
auto& fn_state_update = agent.newFunction("update_agent_state", update_agent_state);  // ���������� ��������� ������
{
    fn_state_update.setMessageOutput("location_agent"); // �������� ������ � �������������� ������-��
    fn_state_update.setAllowAgentDeath(true);
}

// ������� ������ ��������� ��������� ������ � ���������� �� ���������� ������
auto& fn_all_agents = agent.newFunction("density_estimation", density_estimation);
{
    fn_all_agents.setMessageInput("location_agent"); // ��������� ������ � �������������� ������-��
}

model.addInitFunction(init_function);
model.addStepFunction(BasicOutput);
model.addExitCondition(exit_condition); // ����������� ��� ������

{   // Layer #4 ������� �����
    flamegpu::LayerDescription& layer = model.newLayer();
    layer.addHostFunction(agents_data_updating);
}


{   // Layer #1 
    flamegpu::LayerDescription& layer = model.newLayer();
    layer.addAgentFunction(update_agent_state);
}

{   // Layer #2 
    flamegpu::LayerDescription& layer = model.newLayer();
    layer.addAgentFunction(density_estimation);
}

{ // Layer  #3 ������� ������
 flamegpu::LayerDescription& layer = model.newLayer();
 layer.addAgentFunction(agent_move);
}


//�������� �����
flamegpu::EnvironmentDescription& env = model.Environment();
env.newProperty<unsigned int>("DRN", 1);        // ��� �������� �������� ����
env.newProperty<unsigned int>("intensity_of_UGVs", 5); // ������������� �������� ��� � ��� � ������� ���������� �������
env.newProperty<unsigned int>("intensity_of_MGVs", 5); // ������������� �������� ��� � ��� � ������� ���������� �������
env.newProperty<float>("velocity_of_UGVs", 10); // ������� �������� ��� 100 ��/�
env.newProperty<float>("velocity_of_MGVs", 10); // ������� �������� ��� 100 ��/�
env.newProperty<unsigned int>("intensity_of_abnormal_MGVs", 10); // ������������� �������� ���, � ���������� ���������� � ���
env.newProperty<unsigned int>("frequency", 10); // ������� �������� ��� � ��� � ��� (������ N ������)

env.newProperty<unsigned int>("L", 1000); // ����� �����
env.newProperty<unsigned int>("w", 100); // ������ �����
env.newProperty<unsigned int>("N_nodes", 2); // ���������� ������� ���
env.newProperty<unsigned int>("R1", 300); // ���������� ������ ���� ��������� ��������

env.newProperty<unsigned int>("indent_x", 10); // ������� ��� ���
env.newProperty<unsigned int>("indent_y", 10);

env.newMacroProperty<uint32_t>("Traffic"); // ������� ��������� ������ 

/*
* Create Model Runner
*/

flamegpu::CUDASimulation cuda_model(model);
cuda_model.initialise(1, argv);
cuda_model.SimulationConfig().steps = TIME_STOP;
flamegpu::AgentVector population1(model.Agent("agent-vehicles"), 1000);

//��������� ������ ������������ ������ � �������������
if (VIS_MODE == 1)
{
    initVisualisation();
    glutTimerFunc(1, timer, 0);
    cuda_model.SimulationConfig().steps = TIME_STOP;
    std::thread first([&cuda_model]() { cuda_model.simulate(); });
    runVisualisation(); //������ ������������
    first.join();
}


//������������ ������������
if (VIS_MODE == 3)
{
    for (int intensity_of_UGVs = 1; intensity_of_UGVs <= 10; intensity_of_UGVs++)
    {
        for (int velocity_of_UGVs = 1; velocity_of_UGVs <= 10; velocity_of_UGVs++)
        {
            flamegpu::CUDASimulation cuda_model(model);
            cuda_model.initialise(1, argv);

            cuda_model.SimulationConfig().steps = TIME_STOP;
            
            par1 = intensity_of_UGVs;
            par2 = velocity_of_UGVs;
            flamegpu::AgentVector population1(model.Agent("agent-vehicles"), 1000);
            std::thread second([&cuda_model]() { cuda_model.simulate(); });
            second.join();

            //������� �������, ����������� � ������
            int obj = objective.load();

            if (out.is_open())
            {
                out << obj <<
                    ";" << intensity_of_UGVs <<
                    ";" << velocity_of_UGVs << std::endl;
            }

            flamegpu::util::cleanup();

        }
    }
}



out.close();


    return 0;
}
